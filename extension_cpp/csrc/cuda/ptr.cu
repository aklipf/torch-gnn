#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE (256)

namespace extension_cpp
{

  __global__ void ptr_kernel(int numel, const long *n, long *out)
  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numel)
      out[idx] = n[idx] + 1;
  }

  void ptr_out_cuda(const at::Tensor &n, at::Tensor &out)
  {
    TORCH_CHECK(n.sizes() == out.sizes());
    TORCH_CHECK(n.dtype() == at::kLong);
    TORCH_CHECK(out.dtype() == at::kLong);
    TORCH_CHECK(out.is_contiguous());
    TORCH_INTERNAL_ASSERT(n.device().type() == at::DeviceType::CUDA);
    TORCH_INTERNAL_ASSERT(out.device().type() == at::DeviceType::CUDA);
    at::Tensor n_contig = n.contiguous();
    const long *n_ptr = n_contig.data_ptr<long>();
    long *result_ptr = out.data_ptr<long>();
    int numel = n_contig.numel();

    ptr_kernel<<<(numel + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(numel, n_ptr, result_ptr);
  }

  // Registers CUDA implementations for mymuladd, mymul, myadd_out
  TORCH_LIBRARY_IMPL(extension_cpp, CUDA, m)
  {
    m.impl("ptr_out", &ptr_out_cuda);
  }

}
